
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<string.h>
#include<math.h>

//Cuda checks
#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)
inline void _cuda_check(hipError_t ret, const char *file, int line){
    if(ret != hipSuccess) {
        fprintf(stderr, "CudaError: %s %s %d\n", hipGetErrorString(ret), file, line);
        exit(1);
  }
}

#define DIV_ROUND_UP(n, d) (((n) + (d) - 1) / (d))

// Kernal Multiply Function
__global__ void matrix_multiply_kernel(unsigned char *temp, unsigned char *matrix, float *kernal, int order, int middle, int windowSizeX, int windowSizeY){
	//Find place in the execution
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0;
	//If out of bounds, do nothing
	if(y >= windowSizeY || x >= windowSizeX){
		return;
	}
	//Else do function
    for(int y2 = 0; y2 < order; y2++){
		for(int x2 = 0; x2 < order; x2++){
			int tempX = x - middle + x2, tempY = y - middle + y2;
			if(tempX < 0){
				tempX = 0;
			}else if(tempX >= windowSizeX){
				tempX = windowSizeX - 1;
			}
			if(tempY < 0){
				tempY = 0;
			}else if(tempY >= windowSizeY){
				tempY = windowSizeY - 1;
			}
			sum += temp[(windowSizeX * tempY) + tempX] * kernal[(order * x2) + y2];
		}
	}
	//Clamp the sum value
	if(sum < 0){
		sum = 0;
	}else if(sum > 255){
		sum = 255;
	}
	//Add sum value to matrix
	matrix[(windowSizeX * y) + x] = (unsigned char) sum;
        
}

///////////////////
//Multiply Kernal//
///////////////////
void multiplyKernal(unsigned char* matrix, float* kernal, int order, int windowSizeX, int windowSizeY){
	//Find the middle of the kernal
	int middle = ceil(order/2);
	//Declare all the Kernal data
    unsigned char *temp, *matrix_d;
    float *kernal_d;
    int kernal_size = order*order*sizeof(float);
    int matrix_size = windowSizeX * windowSizeY * sizeof(char);
    //Initialize Kernal Data
    cuda_check(hipMalloc(&temp, matrix_size));
    cuda_check(hipMalloc(&matrix_d, matrix_size));
    cuda_check(hipMalloc(&kernal_d, kernal_size));
    // Copy Data to Kernal
    cuda_check(hipMemcpy(temp, matrix, matrix_size, hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(matrix_d, matrix, matrix_size, hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(kernal_d, kernal, kernal_size, hipMemcpyHostToDevice));

    //Kernal Functions
    dim3 block_dim(32, 32);
    dim3 grid_dim(DIV_ROUND_UP(windowSizeX, block_dim.x), DIV_ROUND_UP(windowSizeY, block_dim.y));
    matrix_multiply_kernel<<<grid_dim, block_dim>>>(temp, matrix_d, kernal_d, order, middle, windowSizeX, windowSizeY);
    cuda_check(hipPeekAtLastError());
    cuda_check(hipDeviceSynchronize());

    //Copy back to Host
    cuda_check(hipMemcpy(matrix, matrix_d, matrix_size, hipMemcpyDeviceToHost));
    //Free data
	cuda_check(hipFree(temp));
    cuda_check(hipFree(matrix_d));
    cuda_check(hipFree(kernal_d));
}


/////////////////
//Main Function//
/////////////////
int main(int argc, char **argv)
{
	char firstLine[10];
	int windowSizeX = 0, windowSizeY = 0, temp, order;
	float sigma;

	//Read in inputs, check if they are correct!
	if(argc != 4){
		fprintf(stderr, "Usage: ./gaussian_blur_cuda <input_file> <output_file> <sigma>\n");
		exit(1);
	}
	FILE *fp;
	//If file cannout open
	if((fp = fopen(argv[1], "rb")) == NULL){
		fprintf(stderr, "Error: cannot open file %s\n", argv[1]);
		exit(1);
	}
	// if File is incorrect
	if(fgets(firstLine, 4, fp) == NULL){
		fprintf(stderr, "Error: cannot open file %s\n", argv[1]);
		exit(1);
	}
	//if info is incorrect
	if(strcmp(firstLine, "P5\n")){
		fprintf(stderr, "Error: invalid PGM information\n");
		exit(1);
	}
	
	if(!fscanf(fp,"%d", &windowSizeX) || !fscanf(fp,"%d", &windowSizeY)){
		fprintf(stderr, "Error: cannot open file %s\n", argv[1]);
		exit(1);
	}

	if(windowSizeX == 0 && windowSizeY == 0){
		fprintf(stderr, "Error: invalid PGM information\n");
		exit(1);
	}

	if(!fscanf(fp,"%d", &temp)){
		fprintf(stderr, "Error: cannot open file %s\n", argv[1]);
		exit(1);
	}

	if(temp != 255){
		fprintf(stderr, "Error: invalid PGM information\n");
		exit(1);
	}
	getc(fp);

	//Create matrix
	unsigned char* matrix = (unsigned char*)malloc(sizeof(unsigned char) * windowSizeX * windowSizeY);
	//Read in Matrix
	if(fread(matrix, sizeof(unsigned char), windowSizeX*windowSizeY,fp) != (unsigned)(windowSizeX*windowSizeY)){
		fprintf(stderr, "Error: invalid PGM pixels\n");
		exit(1);
	}

	sigma = atof(argv[3]);
	if(sigma == 0){
		fprintf(stderr, "Error: invalid sigma value\n");
		exit(1);
	}
	//Create the order, if order is even, add 1
	order = ceil(sigma * 6);
	if(order%2 == 0){
		order++;
	}
	if(order > windowSizeX || order > windowSizeY){
		fprintf(stderr, "Error: sigma value too big for image size\n");
		exit(1);
	}
	
	int middle = ceil(order/2);
	//Intialize the kernal
	float sum = 0;
	float kernal [order*order];
	for(int y = 0; y < order; y++){
		for(int x = 0; x < order; x++){
			int x2 = x - middle, y2 = y - middle;
			kernal[(order * y) + x] = (1/(2*M_PI*(pow(sigma,2)))) * (pow(M_E, -((pow(x2,2) + pow(y2,2))/(2*pow(sigma,2)))));
			sum += kernal[(order * y) + x];
		}
	}

	for(int y = 0; y < order; y++){
		for(int x = 0; x < order; x++){
			kernal[(order * y) + x] = kernal[(order * y) + x] / sum;
		}
	}

	//Function Time
    struct timespec before, after;
    clock_gettime(CLOCK_MONOTONIC, &before);
	multiplyKernal(matrix, kernal, order, windowSizeX, windowSizeY);
	clock_gettime(CLOCK_MONOTONIC, &after);
	//Function End
    unsigned long elapsed_ns = (after.tv_sec - before.tv_sec)*(1E9) + after.tv_nsec - before.tv_nsec;
    double seconds = elapsed_ns / (1E9);

	printf("Running time: %f secs\n", seconds);


	char name[255];
	sprintf(name, "%s", argv[2]);
    
	//Print back to output file
    FILE *fd;
    fd = fopen(name, "w+");
    fprintf(fd, "P5\n");
    fprintf(fd, "%d %d\n", windowSizeX, windowSizeY);
    fprintf(fd, "255\n");

    for(int y = 0; y < windowSizeY; y++) {
        for(int x = 0; x < windowSizeX; x++) {
            fprintf(fd, "%c", matrix[(windowSizeX * y) + x]);
        }
    }

	fclose(fp);
	fclose(fd);
	free(matrix);

	return 0;
}